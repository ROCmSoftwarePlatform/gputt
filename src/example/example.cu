#include "hip/hip_runtime.h"
#include <gputt.h>
#include <gputt_internal.h>

#include <iostream>
#include <vector>

//
// Error checking wrapper for gpuTT and vendor API.
//

#define GPUTT_ERR_CHECK(stmt)                                                  \
  do {                                                                         \
    gputtResult err = stmt;                                                    \
    if (err != GPUTT_SUCCESS) {                                                \
      fprintf(stderr, "Error \"%d\" at %s :%d\n", err, __FILE__, __LINE__);    \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)

#define GPU_ERR_CHECK(x)                                                       \
  do {                                                                         \
    gpuError_t err = x;                                                        \
    if (err != gpuSuccess) {                                                   \
      fprintf(stderr, "Error \"%s\" at %s :%d \n", gpuGetErrorString(err),     \
              __FILE__, __LINE__);                                             \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)

__host__
inline bool operator!=(const __half& x, const __half& y)
{
  return memcmp(&x, &y, sizeof(__half));
}

inline std::ostream& operator<<(std::ostream& os, __half& val)
{
  os << static_cast<double>(val);
  return os;
}

inline std::ostream& operator<<(std::ostream& os, char4& val)
{
  os << static_cast<int>(val.x) << " " << static_cast<int>(val.y) << " " <<
    static_cast<int>(val.z) << " " << static_cast<int>(val.w);
  return os;
}

inline std::ostream& operator<<(std::ostream& os, uchar4& val)
{
  os << static_cast<int>(val.x) << " " << static_cast<int>(val.y) << " " <<
    static_cast<int>(val.z) << " " << static_cast<int>(val.w);
  return os;
}

template <typename D, typename T>
static void check(D &dim, T &idata, T &odata) {
  // Perform the same permutation on the CPU.
  T odata2(odata.size());
  for (int d0 = 0; d0 < dim[0]; d0++)
    for (int d1 = 0; d1 < dim[1]; d1++)
      for (int d2 = 0; d2 < dim[2]; d2++)
        for (int d3 = 0; d3 < dim[3]; d3++) {
          auto in = idata[d3 * dim[2] * dim[1] * dim[0] + d2 * dim[1] * dim[0] +
                          d1 * dim[0] + d0];

          // int permutation[4] = {3, 0, 2, 1};
          auto &out2 = odata2[d1 * dim[2] * dim[0] * dim[3] +
                              d2 * dim[0] * dim[3] + d0 * dim[3] + d3];

          out2 = in;

          // Compare with gpuTT's output element.
          auto out = odata[d1 * dim[2] * dim[0] * dim[3] +
                           d2 * dim[0] * dim[3] + d0 * dim[3] + d3];
#if 1
          if (out != out2) {
            std::cout << "Output elements mismatch at [" << d0 << "][" << d1
                      << "][" << d2 << "][" << d3 << "]: " << out
                      << " != " << out2 << std::endl;
            exit(-1);
          }
#endif
        }

  if (memcmp(odata.data(), odata2.data(), odata.size() * sizeof(odata[0]))) {
    fprintf(stderr, "Output tensors mismatch\n");
#if 1
    exit(-1);
#endif
  }
}

template <typename T> static void test() {
  std::cout << "Testing type " << gputtGetDataTypeString(gputtGetDataType<T>()) <<
    " of size " << sizeof(T) << std::endl;
  
  // Four dimensional tensor
//#define LARGE_SIZE
#ifdef LARGE_SIZE
  int dim[4] = { 101, 103, 107, 109 };
#else
  // Transpose (31, 549, 2, 3) -> (3, 31, 2, 549)
  int dim[4] = { 31, 549, 2, 3 };
#endif
  int permutation[4] = { 3, 0, 2, 1 };

  std::vector<T> idata(dim[0] * dim[1] * dim[2] * dim[3]);
  for (int i = 0; i < idata.size(); i++)
    idata[i] = T(i % 2);
  std::vector<T> odata(idata.size());

  gputtHandle plan;
  std::vector<gputtHandle> plans;

  // Option 1: Create plan on NULL stream and choose the method manually.
  for (int i = 0; i < NumTransposeMethods; i++) {
    auto method = static_cast<gputtTransposeMethod>(i);

    // Only use the methods that are supported for the given parameters.
    if (GPUTT_SUCCESS ==
        gputtPlan(&plan, 4, dim, permutation, gputtGetDataType<T>(), 0, method))
      plans.push_back(plan);
  }

  // Option 2: Create plan on NULL stream and choose the method based on
  // heuristics GPUTT_ERR_CHECK(gputtPlan(&plan, 4, dim, permutation,
  // sizeof(idata[0]), 0)); plans.push_back(plan);

  // Option 3: Create plan on NULL stream and choose the method based on
  // performance measurements GPUTT_ERR_CHECK(gputtPlanMeasure(&plan, 4, dim,
  // permutation, sizeof(idata[0]), 0, idata, odata)); plans.push_back(plan);

  for (auto plan : plans) {
    T *idataGPU;
    GPU_ERR_CHECK(gpuMalloc(&idataGPU, idata.size() * sizeof(idata[0])));
    GPU_ERR_CHECK(gpuMemcpy(idataGPU, idata.data(),
                            idata.size() * sizeof(idata[0]),
                            gpuMemcpyHostToDevice));

    T *odataGPU;
    GPU_ERR_CHECK(gpuMalloc(&odataGPU, odata.size() * sizeof(odata[0])));

    gputtTransposeMethod method;
    GPUTT_ERR_CHECK(gputtPlanMethod(plan, &method));
    std::cout << "Testing method " << gputtGetTransposeMethodString(method);

    // Execute plan
    auto start = std::chrono::high_resolution_clock::now();
    GPUTT_ERR_CHECK(gputtExecute(plan, idataGPU, odataGPU));

    GPU_ERR_CHECK(gpuDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();

    std::cout << ": time = " << std::chrono::duration_cast<std::chrono::duration<double>>(
      end - start).count() << " sec" << std::endl;

    GPU_ERR_CHECK(gpuMemcpy(odata.data(), odataGPU,
                            odata.size() * sizeof(odata[0]),
                            gpuMemcpyDeviceToHost));

    // Destroy plan
    GPUTT_ERR_CHECK(gputtDestroy(plan));

    GPU_ERR_CHECK(gpuFree(idataGPU));
    GPU_ERR_CHECK(gpuFree(odataGPU));

    check(dim, idata, odata);
  }
}

int main(int argc, char *argv[]) {
  test<  double>();
  test<   float>();
  test<  __half>();
  test< int64_t>();
  test<uint64_t>();
  test< int32_t>();
  test<uint32_t>();
  test< int16_t>();
  test<uint16_t>();
  test<  int8_t>();
  test< uint8_t>();
  test<   char4>();
  test<  uchar4>();
  return 0;
}
